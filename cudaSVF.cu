#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

void
CudaRenderer::render() {

    int size = imageWidth * imageHeight;

    // 256 threads per block is a healthy number
    dim3 blockDim(THREADS, 1);
    dim3 gridDim((size + blockDim.x - 1) / blockDim.x);

    if (sceneName == SNOWFLAKES || sceneName == SNOWFLAKES_SINGLE_FRAME)
        kernelRenderCircles<true><<<gridDim, blockDim>>>();
    else
        kernelRenderCircles<false><<<gridDim, blockDim>>>();

    hipDeviceSynchronize();
}

void main()
{
}
