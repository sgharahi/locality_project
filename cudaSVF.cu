#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#define ABS(x) (x > 0 ? x : -x)
#define MIN(x,y) (x < y ? x : y)

__global void SVF()
{



}

__device int DTW(device_vector<int>& interval_i, device_vector<int>& interval_j)
{
    device_vector<device_vector<int>> dtw(interval_i.size());
    for(int i = 0 ; i < dtw.size(); i++)
    {
        dtw[i] = device_vector<int>(interval_j.size());
    }
    
    for(int i = 0; i < dtw.size(); i++)
    {
        dtw[i][0] = (1 << 31) - 1;
        dtw[0][i] = (1 << 31) - 1;
    }

    dtw[0][0] = 0;
    
    for(i = 1; i < dtw.size(); i++)
    {
        for(j = 1; j < dtw[i].size(); j++)
        {
            cost = ABS(interval_i[i] - interval_j[j]);
            dtw[i][j] = cost + MIN(dtw[i-1][j], MIN(dtw[i][j-1], dtw[i-1][j-1]));
        }
    }

    return dtw[interval_i.size() - 1][interval_j.size() - 1];
}

__global void simMatrix(device_vector<device_vector<int>>& v_trace, device_vector<int>& a_trace, int max_len)
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    int i = idx % max_len, j = idx / max_len;

    a_sim[i][j] = ABS(a_trace[i] - a_trace[j]);

    v_sim[i][j] = DTW(v_trace[i], v_trace[j]);
}

void cudaSVF(device_vector<device_vector<int>>& v_trace, device_vector<int>& a_trace)
{
    int max_len = v_trace.size() > a_trace.size() ? v_trace.size() : a_trace.size();
    int size = max_len * max_len;

    // 1024 threads per block
    dim3 blockDim(1024, 1);
    dim3 gridDim((size + blockDim.x - 1) / blockDim.x);

    simMatrix<<<gridDim, blockDim>>>(v_trace, a_trace, max_len);

    hipDeviceSynchronize();
}
